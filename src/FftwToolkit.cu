#include "hip/hip_runtime.h"
#include "FftwToolkit.h"
struct MallocDeleter {
    void operator()(hipfftDoubleReal* ptr) const {
        free(ptr);
    }
    void operator()(hipfftDoubleComplex* ptr) const {
        free(ptr);
    }
};
__global__ void hatConvCalculate(hipfftDoubleReal *Rsrc1,hipfftDoubleReal *Rsrc2,int n)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx<n)
	{
		Rsrc1[idx] = Rsrc1[idx]*Rsrc2[idx];
	}
}
void hatConv(scftData* sc1,hipfftDoubleComplex *rslt, hipfftDoubleComplex *src1, hipfftDoubleComplex *src2)
{
    
    checkCudaErrors(hipMemset(sc1->realGpu3,0,sizeof(hipfftDoubleReal)*sc1->realDofs));
    checkCudaErrors(hipMemset(sc1->realGpu2,0,sizeof(hipfftDoubleReal)*sc1->realDofs));
    FftwC2R(src1,sc1->realGpu3,sc1);
    FftwC2R(src2,sc1->realGpu2,sc1);
	hatConvCalculate<<<(sc1->realDofs-1)/1024+1,1024>>>(sc1->realGpu3,sc1->realGpu2,sc1->realDofs);
    FftwR2C(sc1->realGpu3,rslt,sc1);

}
double Intergral_space(scftData* sc1,hipfftDoubleComplex *src1, hipfftDoubleComplex *src2)
{
    // hipfftDoubleReal *Rsrc1,*Rsrc2;
    // hipfftDoubleComplex *rslt,*rsltC;
    // checkCudaErrors(hipMalloc((void**)&Rsrc1,sizeof(hipfftDoubleReal)*sc1->realDofs));
    // checkCudaErrors(hipMemset(Rsrc1,0,sizeof(hipfftDoubleReal)*sc1->realDofs));
    // checkCudaErrors(hipMalloc((void**)&Rsrc2,sizeof(hipfftDoubleReal)*sc1->realDofs));
    // checkCudaErrors(hipMemset(Rsrc2,0,sizeof(hipfftDoubleReal)*sc1->realDofs));
    // checkCudaErrors(hipMalloc((void**)&rslt,sizeof(hipfftDoubleComplex)*sc1->cplxDofs));
    // checkCudaErrors(hipMemset(rslt,0,sizeof(hipfftDoubleComplex)*sc1->cplxDofs));
    // rsltC=(hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*1);
    // memset(rsltC,0,sizeof(hipfftDoubleComplex)*1);

    checkCudaErrors(hipMemset(sc1->realGpu3,0,sizeof(hipfftDoubleReal)*sc1->realDofs));
    checkCudaErrors(hipMemset(sc1->realGpu2,0,sizeof(hipfftDoubleReal)*sc1->realDofs));
    checkCudaErrors(hipMemset(sc1->cplxGpu2,0,sizeof(hipfftDoubleComplex)*sc1->cplxDofs));
    memset(sc1->cplxC1,0,sizeof(hipfftDoubleComplex)*1);
    FftwC2R(src1,sc1->realGpu3,sc1);
    FftwC2R(src2,sc1->realGpu2,sc1);
	hatConvCalculate<<<(sc1->realDofs-1)/1024+1,1024>>>(sc1->realGpu3,sc1->realGpu2,sc1->realDofs);
    FftwR2C(sc1->realGpu3,sc1->cplxGpu2,sc1);
    checkCudaErrors(hipMemcpy(sc1->cplxC1,sc1->cplxGpu2,sizeof(hipfftDoubleComplex)*1,hipMemcpyDeviceToHost));
    double val=sc1->cplxC1[0].x;
    // checkCudaErrors(hipFree(rslt));
    // free(rsltC);
	// checkCudaErrors(hipFree(Rsrc1));
	// checkCudaErrors(hipFree(Rsrc2));
    return val;
    
}

// void FftwC2R(hipfftDoubleComplex *Corig,hipfftDoubleReal *Rrslt, scftData* sc1)
// {
  

//     // hipfftDoubleComplex *Q_Ctmp;/sc1->cplxGpu1
//     // checkCudaErrors(hipMalloc((void**)&Q_Ctmp,sizeof(hipfftDoubleComplex)*sc1->cplxDofs));
//     // checkCudaErrors(hipMemset(Q_Ctmp,0,sizeof(hipfftDoubleComplex)*sc1->cplxDofs));
//     // hipfftDoubleReal *Q_Rtmp;
//     // checkCudaErrors(hipMalloc((void**)&Q_Rtmp,sizeof(hipfftDoubleReal)*sc1->realDofs));
//     // checkCudaErrors(hipMemset(Q_Rtmp,0,sizeof(hipfftDoubleReal)*sc1->realDofs));
//     checkCudaErrors(hipMemset(sc1->cplxGpu1,0,sizeof(hipfftDoubleComplex)*sc1->cplxDofs));
//     checkCudaErrors(hipMemcpy(sc1->cplxGpu1,Corig,sizeof(hipfftDoubleComplex)*sc1->cplxDofs,hipMemcpyDeviceToDevice));
//     checkCudaErrors(hipfftExecZ2D(sc1->plan,sc1->cplxGpu1,Rrslt));
//     // checkCudaErrors(hipFree(Q_Ctmp));
//     // checkCudaErrors(hipFree(Q_Rtmp));
// }
void FftwC2R(hipfftDoubleComplex *Corig,hipfftDoubleReal *Rrslt, scftData* sc1)
{


    // hipfftDoubleComplex *Q_Ctmp;/sc1->cplxGpu1
    // checkCudaErrors(hipMalloc((void**)&Q_Ctmp,sizeof(hipfftDoubleComplex)*sc1->cplxDofs));
    // checkCudaErrors(hipMemset(Q_Ctmp,0,sizeof(hipfftDoubleComplex)*sc1->cplxDofs));
    // hipfftDoubleReal *Q_Rtmp;
    // checkCudaErrors(hipMalloc((void**)&Q_Rtmp,sizeof(hipfftDoubleReal)*sc1->realDofs));
    // checkCudaErrors(hipMemset(Q_Rtmp,0,sizeof(hipfftDoubleReal)*sc1->realDofs));
    checkCudaErrors(hipMemset(sc1->cplxGpu1,0,sizeof(hipfftDoubleComplex)*sc1->cplxDofs));
    checkCudaErrors(hipMemcpy(sc1->cplxGpu1,Corig,sizeof(hipfftDoubleComplex)*sc1->cplxDofs,hipMemcpyDeviceToDevice));
    checkCudaErrors(hipfftExecZ2D(sc1->plan,sc1->cplxGpu1,Rrslt));
    // checkCudaErrors(hipMemcpy(Rrslt,sc1->realGpu1,sizeof(hipfftDoubleReal)*sc1->realDofs,hipMemcpyDeviceToDevice));
    // checkCudaErrors(hipFree(Q_Ctmp));
    // checkCudaErrors(hipFree(Q_Rtmp));
}

void FftwR2C(hipfftDoubleReal *Rrslt, hipfftDoubleComplex *Corig,scftData* sc1)
{
    
    double realDofs1=1.0/(double)sc1->realDofs;

    checkCudaErrors(hipfftExecD2Z(sc1->plan1,Rrslt,Corig));
   
    checkCudaErrors(hipblasZdscal(sc1->handle,sc1->cplxDofs,&realDofs1,Corig,1));
  
    // checkCudaErrors(hipMemcpy(Corig,sc1->cplxGpu1,sizeof(hipfftDoubleComplex)*sc1->cplxDofs,hipMemcpyDeviceToDevice));
    // checkCudaErrors(hipFree(Q_Ctmp));
    // checkCudaErrors(hipFree(Q_Rtmp));
   
}


